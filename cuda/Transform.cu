#include "hip/hip_runtime.h"
#include "cuda.cuh"


__device__ void RotatePoint(float *point, float* R, float* T)
{
    if(point[0]==0 && point[1]==0 && point[2]==0) {
        return;
    }
	float res[3];
	point[0] += T[0];
	point[1] += T[1];
	point[2] += T[2];

	res[0] = point[0] * R[0] + point[1] * R[1] + point[2] * R[2];
	res[1] = point[0] * R[3] + point[1] * R[4] + point[2] * R[5];
	res[2] = point[0] * R[6] + point[1] * R[7] + point[2] * R[8];

	point[0] = res[0];
	point[1] = res[1];
	point[2] = res[2];
}


__global__ void Transform_Kernel(int width_d, int height_d, Point3fRGB *vertices, float *R, float* T)
{
    float tempPoint[3];

    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int ptr_idx = i*width_d + j;

    tempPoint[0] = vertices[ptr_idx].X;
    tempPoint[1] = vertices[ptr_idx].Y;
    tempPoint[2] = vertices[ptr_idx].Z;

    RotatePoint(tempPoint, R, T);

    vertices[ptr_idx].X = tempPoint[0];
    vertices[ptr_idx].Y = tempPoint[1];
    vertices[ptr_idx].Z = tempPoint[2];
}


TransformStruct* Transform_gpu_init()
{
    TransformStruct *transformStruct;
    hipMallocManaged((void**)&transformStruct, sizeof(TransformStruct));
    hipMalloc((void**)&transformStruct->dev_vertices, sizeof(Point3fRGB) * Width_depth_HR * Height_depth_HR);
    hipMalloc((void**)&transformStruct->dev_R, sizeof(float) * 9);
    hipMalloc((void**)&transformStruct->dev_T, sizeof(float) * 3);
    return transformStruct;
}


void Transform(int width_d, int height_d, framePacket *packet, std::vector<std::vector<float>> &R, std::vector<float> &T, TransformStruct *transformStruct)
{
#if 0
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
#endif
    hipMemcpy(transformStruct->dev_vertices, packet->vertices, width_d * height_d * sizeof(Point3fRGB), hipMemcpyHostToDevice );

    float host_R[9];
    float host_T[3];

    host_R[0] = R[0][0];
    host_R[1] = R[0][1];
    host_R[2] = R[0][2];
    host_R[3] = R[1][0];
    host_R[4] = R[1][1];
    host_R[5] = R[1][2];
    host_R[6] = R[2][0];
    host_R[7] = R[2][1];
    host_R[8] = R[2][2];
    host_T[0] = T[0];
    host_T[1] = T[1];
    host_T[2] = T[2];

    hipMemcpy(transformStruct->dev_R, host_R, 9 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(transformStruct->dev_T, host_T, 3 * sizeof(float), hipMemcpyHostToDevice);

    dim3 blocks(width_d / 8, height_d / 8);
    dim3 threads(8, 8);

    Transform_Kernel<<<blocks, threads>>>(width_d, height_d, transformStruct->dev_vertices, transformStruct->dev_R, transformStruct->dev_T);

    hipMemcpy(packet->vertices, transformStruct->dev_vertices, width_d * height_d * sizeof(Point3fRGB), hipMemcpyDeviceToHost);

#if 0
    hipEventRecord(end);
    hipEventSynchronize(end);
    float millisecond = 0;
    hipEventElapsedTime(&millisecond, start, end);
    printf("\t transform time = %f ms\n", millisecond);
#endif
}
