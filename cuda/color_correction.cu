#include "hip/hip_runtime.h"
#include "cuda.cuh"


extern __device__ void backforward_map(float *point, float *R, float *T)
{
    float res[3];
    res[0] = point[0] * R[0] + point[1] * R[1] + point[2] * R[2];
    res[1] = point[0] * R[3] + point[1] * R[4] + point[2] * R[5];
    res[2] = point[0] * R[6] + point[1] * R[7] + point[2] * R[8];

    point[0] = res[0] - T[0];
    point[1] = res[1] - T[1];
    point[2] = res[2] - T[2];
}


__global__ void get_matched_points_kernel(Context_gpu* ctx_gpu, Point3fRGB* left, Point3fRGB* right, float *depth_right, int idx) {
    int x_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int y_idx = threadIdx.y + blockIdx.y * blockDim.y;
    int pix_idx = x_idx + y_idx * blockDim.x * gridDim.x;

    float tempPoint[3];
    tempPoint[0] = left[pix_idx].X;
    tempPoint[1] = left[pix_idx].Y;
    tempPoint[2] = left[pix_idx].Z;

    backforward_map(tempPoint, ctx_gpu->invR[idx], ctx_gpu->T[idx]);

    if(tempPoint[2] <= 0) {
        return;
    }

    int x = (int)(tempPoint[0] * ctx_gpu->x_ratio / tempPoint[2] * ctx_gpu->K[idx].fx + ctx_gpu->K[idx].cx - 0.5);
    int y = (int)(tempPoint[1] * ctx_gpu->y_ratio / tempPoint[2] * ctx_gpu->K[idx].fy + ctx_gpu->K[idx].cy - 0.5);

    int index;

    if(x >= 0 && x < Width_depth_HR && y >= 0 && y < Height_depth_HR) {
        for(int i=0; i<patch_size; i++) {
            for(int j=0; j<patch_size; j++) {
                index = ((y - patch_size/2) + i) * Width_depth_HR + (x - patch_size/2) + j;
                if(index >= 0 && index < Width_depth_HR * Height_depth_HR && depth_right[index] > 0 && abs(tempPoint[2] * 1000 - depth_right[index]) < 20) {
                    ctx_gpu->matched_idx[pix_idx] = y * Width_depth_HR + x;
                }
            }
        }
    }

    return;
}


void get_matched_points_cuda(Context_gpu* ctx_gpu, Point3fRGB *verts1, Point3fRGB *verts2, float *depth_right, int cam_idx) {
    hipMemcpy(ctx_gpu->verts1_match, verts1, sizeof(Point3fRGB) * Width_depth_HR * Height_depth_HR, hipMemcpyHostToDevice);
    hipMemcpy(ctx_gpu->verts2_match, verts2, sizeof(Point3fRGB) * Width_depth_HR * Height_depth_HR, hipMemcpyHostToDevice);
    hipMemcpy(ctx_gpu->depth_match, depth_right, sizeof(float) * Width_depth_HR * Height_depth_HR, hipMemcpyHostToDevice);

    dim3 blocks(Width_depth_HR / 32, Height_depth_HR / 8);
    dim3 threads(32, 8);

    get_matched_points_kernel<<<blocks, threads>>>(ctx_gpu, ctx_gpu->verts1_match, ctx_gpu->verts2_match, ctx_gpu->depth_match, cam_idx);

    int idx = 0;
    FILE *f = fopen("match_points_warp.csv", "w");
    for(int j=0; j<Width_depth_HR * Height_depth_HR; j++) {
        idx = ctx_gpu->matched_idx[j];
        if(idx > 0 && idx < Width_depth_HR * Height_depth_HR) {
            fprintf(f, "%f,%f,%f,%f,%f,%f,%u,%u,%u,%u,%u,%u\n", ctx_gpu->verts1_match[j].X, ctx_gpu->verts1_match[j].Y, ctx_gpu->verts1_match[j].Z, 
                                                                ctx_gpu->verts2_match[idx].X, ctx_gpu->verts2_match[idx].Y, ctx_gpu->verts2_match[idx].Z,
                                                                ctx_gpu->verts1_match[j].R, ctx_gpu->verts1_match[j].G, ctx_gpu->verts1_match[j].B, 
                                                                ctx_gpu->verts2_match[idx].R, ctx_gpu->verts2_match[idx].G, ctx_gpu->verts2_match[idx].B);
        }
    }
    fclose(f);
}