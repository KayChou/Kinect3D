#include "hip/hip_runtime.h"
#include "cuda.cuh"

#define patch_size 4

Context_gpu* create_context(Context* ctx_cpu) {
    hipSetDevice(0);

    Context_gpu *ctx_gpu;
    hipMallocManaged((void**)&ctx_gpu, sizeof(Context_gpu));
    hipMallocManaged((void**)&ctx_gpu->depth_out, 512 * 424 * sizeof(float));

    for(int i=0; i<numKinects; i++) {
        hipMallocManaged((void**)&ctx_gpu->R[i], sizeof(float) * 9);
        hipMallocManaged((void**)&ctx_gpu->T[i], sizeof(float) * 3);
        hipMallocManaged((void**)&ctx_gpu->invR[i], sizeof(float) * 9);
        hipMalloc((void**)&ctx_gpu->vertices[i], sizeof(Point3fRGB) * Width_depth_HR * Height_depth_HR);
        hipMalloc((void**)&ctx_gpu->depth[i], sizeof(float) * Width_depth_HR * Height_depth_HR);
    }
    ctx_gpu->width = Width_depth_HR;
    ctx_gpu->height = Height_depth_HR;
    ctx_gpu->x_ratio = Width_depth_HR / 512.f;
    ctx_gpu->y_ratio = Height_depth_HR / 424.f;
    
    return ctx_gpu;
}


void updata_context(Context_gpu *ctx_gpu, Context *ctx_cpu) {
    for(int i=0; i<numKinects; i++) {
        hipMemcpy(&ctx_gpu->R[i][0], &ctx_cpu->R[i][0][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][1], &ctx_cpu->R[i][0][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][2], &ctx_cpu->R[i][0][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][3], &ctx_cpu->R[i][1][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][4], &ctx_cpu->R[i][1][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][5], &ctx_cpu->R[i][1][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][6], &ctx_cpu->R[i][2][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][7], &ctx_cpu->R[i][2][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][8], &ctx_cpu->R[i][2][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->T[i][0], &ctx_cpu->T[i][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->T[i][1], &ctx_cpu->T[i][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->T[i][2], &ctx_cpu->T[i][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][0], &ctx_cpu->invR[i][0][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][1], &ctx_cpu->invR[i][0][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][2], &ctx_cpu->invR[i][0][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][3], &ctx_cpu->invR[i][1][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][4], &ctx_cpu->invR[i][1][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][5], &ctx_cpu->invR[i][1][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][6], &ctx_cpu->invR[i][2][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][7], &ctx_cpu->invR[i][2][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][8], &ctx_cpu->invR[i][2][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->K[i].fx, &ctx_cpu->K[i].fx, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->K[i].fy, &ctx_cpu->K[i].fy, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->K[i].cx, &ctx_cpu->K[i].cx, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->K[i].cy, &ctx_cpu->K[i].cy, sizeof(float), hipMemcpyHostToDevice);
    }
}


__device__ void backforward_mapping(float *point, float *R, float *T)
{
    float res[3];
    res[0] = point[0] * R[0] + point[1] * R[1] + point[2] * R[2];
    res[1] = point[0] * R[3] + point[1] * R[4] + point[2] * R[5];
    res[2] = point[0] * R[6] + point[1] * R[7] + point[2] * R[8];

    point[0] = res[0] - T[0];
    point[1] = res[1] - T[1];
    point[2] = res[2] - T[2];
}


__global__ void overlap_removal_kernel(Context_gpu* ctx_gpu, Point3fRGB* left, float* depth_left, Point3fRGB* right, float* depth_right, int idx, float *depth_out) {
    int x_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int y_idx = threadIdx.y + blockIdx.y * blockDim.y;
    int pix_idx = x_idx + y_idx * blockDim.x * gridDim.x;
    depth_out[pix_idx] = 128;

    float tempPoint[3];
    tempPoint[0] = left[pix_idx].X;
    tempPoint[1] = left[pix_idx].Y;
    tempPoint[2] = left[pix_idx].Z;

    backforward_mapping(tempPoint, ctx_gpu->invR[idx], ctx_gpu->T[idx]);

    if(tempPoint[2] <= 0) {
        return;
    }

    int x = (int)(tempPoint[0] * ctx_gpu->x_ratio / tempPoint[2] * ctx_gpu->K[idx].fx + ctx_gpu->K[idx].cx - 0.5);
    int y = (int)(tempPoint[1] * ctx_gpu->y_ratio / tempPoint[2] * ctx_gpu->K[idx].fy + ctx_gpu->K[idx].cy - 0.5);

    bool b_overlap = false;
    int index;

    if(x >= 0 && x < Width_depth_HR && y >= 0 && y < Height_depth_HR) {
        for(int i=0; i<patch_size; i++) {
            for(int j=0; j<patch_size; j++) {
                index = ((y - patch_size/2) + i) * Width_depth_HR + (x - patch_size/2) + j;
                if(index >= 0 && index < Width_depth_HR * Height_depth_HR && depth_right[index] > 0 && abs(tempPoint[2] * 1000 - depth_right[index]) < 20) {
                    b_overlap = true;
                }
            }
        }
    }

    if(b_overlap) {
        left[pix_idx].X = 0;
        left[pix_idx].Y = 0;
        left[pix_idx].Z = 0;
        depth_left[pix_idx] = 0;
    }
}


void overlap_removal_cuda(Context_gpu* ctx_gpu, framePacket** frameList, float* dpeth_out) {
#if 0
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
#endif
    for(int i=0; i<numKinects; i++) {
        hipMemcpy(ctx_gpu->vertices[i], frameList[i]->vertices, sizeof(Point3fRGB) * Width_depth_HR * Height_depth_HR, hipMemcpyHostToDevice);
        hipMemcpy(ctx_gpu->depth[i], frameList[i]->data_d, sizeof(float) * Width_depth_HR * Height_depth_HR, hipMemcpyHostToDevice);
    }

    dim3 blocks(Width_depth_HR / 32, Height_depth_HR / 8);
    dim3 threads(32, 8);

    int idx;

    for(int i=0; i<numKinects; i++) {
        idx = (i + 1) % numKinects;
        overlap_removal_kernel<<<blocks, threads>>>(ctx_gpu, ctx_gpu->vertices[i], ctx_gpu->depth[i], ctx_gpu->vertices[idx], ctx_gpu->depth[idx], idx, ctx_gpu->depth_out);
    }

    for(int i=0; i<numKinects; i++) {
        hipMemcpy(frameList[i]->vertices, ctx_gpu->vertices[i], sizeof(Point3fRGB) * Width_depth_HR * Height_depth_HR, hipMemcpyDeviceToHost);
    }
    hipMemcpy(dpeth_out, ctx_gpu->depth_out, sizeof(float) * 512 * 424, hipMemcpyDeviceToHost);

#if 0
    hipEventRecord(end);
    hipEventSynchronize(end);
    float millisecond = 0;
    hipEventElapsedTime(&millisecond, start, end);
    printf("\t overlap_removal_cuda time = %f ms\n", millisecond);
#endif
}