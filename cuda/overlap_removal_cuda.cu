#include "hip/hip_runtime.h"
#include "cuda.cuh"

#define patch_size 8

Context_gpu* create_context(Context* ctx_cpu) {
    hipSetDevice(0);

    Context_gpu *ctx_gpu;
    hipMallocManaged((void**)&ctx_gpu, sizeof(Context_gpu));

    for(int i=0; i<numKinects; i++) {
        hipMallocManaged((void**)&ctx_gpu->R[i], sizeof(float) * 9);
        hipMallocManaged((void**)&ctx_gpu->T[i], sizeof(float) * 3);
        hipMallocManaged((void**)&ctx_gpu->invR[i], sizeof(float) * 9);
        hipMalloc((void**)&ctx_gpu->vertices[i], sizeof(Point3fRGB) * Width_depth_HR * Height_depth_HR);
        hipMalloc((void**)&ctx_gpu->depth[i], sizeof(float) * Width_depth_HR * Height_depth_HR);
        hipMalloc((void**)&ctx_gpu->vertices_shift[i], sizeof(Point3fRGB) * Width_depth_HR * Height_depth_HR);
        hipMalloc((void**)&ctx_gpu->depth_shift[i], sizeof(float) * Width_depth_HR * Height_depth_HR);
    }
    ctx_gpu->width = Width_depth_HR;
    ctx_gpu->height = Height_depth_HR;
    ctx_gpu->x_ratio = Width_depth_HR / 512.f;
    ctx_gpu->y_ratio = Height_depth_HR / 424.f;
    
    return ctx_gpu;
}


void updata_context(Context_gpu *ctx_gpu, Context *ctx_cpu) {
    for(int i=0; i<numKinects; i++) {
        hipMemcpy(&ctx_gpu->R[i][0], &ctx_cpu->R[i][0][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][1], &ctx_cpu->R[i][0][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][2], &ctx_cpu->R[i][0][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][3], &ctx_cpu->R[i][1][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][4], &ctx_cpu->R[i][1][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][5], &ctx_cpu->R[i][1][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][6], &ctx_cpu->R[i][2][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][7], &ctx_cpu->R[i][2][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][8], &ctx_cpu->R[i][2][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->T[i][0], &ctx_cpu->T[i][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->T[i][1], &ctx_cpu->T[i][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->T[i][2], &ctx_cpu->T[i][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][0], &ctx_cpu->invR[i][0][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][1], &ctx_cpu->invR[i][0][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][2], &ctx_cpu->invR[i][0][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][3], &ctx_cpu->invR[i][1][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][4], &ctx_cpu->invR[i][1][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][5], &ctx_cpu->invR[i][1][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][6], &ctx_cpu->invR[i][2][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][7], &ctx_cpu->invR[i][2][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][8], &ctx_cpu->invR[i][2][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->K[i].fx, &ctx_cpu->K[i].fx, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->K[i].fy, &ctx_cpu->K[i].fy, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->K[i].cx, &ctx_cpu->K[i].cx, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->K[i].cy, &ctx_cpu->K[i].cy, sizeof(float), hipMemcpyHostToDevice);
    }
}


__device__ void backforward_mapping(float *point, float *R, float *T)
{
    float res[3];
    res[0] = point[0] * R[0] + point[1] * R[1] + point[2] * R[2];
    res[1] = point[0] * R[3] + point[1] * R[4] + point[2] * R[5];
    res[2] = point[0] * R[6] + point[1] * R[7] + point[2] * R[8];

    point[0] = res[0] - T[0];
    point[1] = res[1] - T[1];
    point[2] = res[2] - T[2];
}


__global__ void overlap_removal_kernel(Context_gpu* ctx_gpu, Point3fRGB* left, Point3fRGB* right, float* depth, int idx) {
    int x_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int y_idx = threadIdx.y + blockIdx.y * blockDim.y;
    int pix_idx = x_idx + y_idx * blockDim.x * gridDim.x;

    float tempPoint[3];
    tempPoint[0] = left[pix_idx].X;
    tempPoint[1] = left[pix_idx].Y;
    tempPoint[2] = left[pix_idx].Z;

    backforward_mapping(tempPoint, ctx_gpu->invR[idx], ctx_gpu->T[idx]);

    if(tempPoint[2] <= 0) {
        return;
    }

    int x = (int)(tempPoint[0] * ctx_gpu->x_ratio / tempPoint[2] * ctx_gpu->K[idx].fx + ctx_gpu->K[idx].cx - 0.5);
    int y = (int)(tempPoint[1] * ctx_gpu->y_ratio / tempPoint[2] * ctx_gpu->K[idx].fy + ctx_gpu->K[idx].cy - 0.5);

    bool flag = false;
    int index;

    if(x >= 0 && x < Width_depth_HR && y >= 0 && y < Height_depth_HR) {
        for(int i=0; i<patch_size; i++) {
            for(int j=0; j<patch_size; j++) {
                index = ((y - patch_size/2) + i) * Width_depth_HR + (x - patch_size/2) + j;
                if(index >= 0 && index < Width_depth_HR * Height_depth_HR && tempPoint[2] > (depth[index] + 0.1)) {
                    flag = true;
                }
            }
        }
    }

    if(flag) {
        // if(idx == 0) {
        //     right[pix_idx].R = 255;
        //     right[pix_idx].G = 0;
        //     right[pix_idx].B = 0;
        // }
        // if(idx == 1) {
        //     right[pix_idx].R = 0;
        //     right[pix_idx].G = 255;
        //     right[pix_idx].B = 0;
        // }
        // right[pix_idx].X = 0;
        // right[pix_idx].Y = 0;
        // right[pix_idx].Z = 0;
        right[pix_idx].R = 255;
        right[pix_idx].G = 0;
        right[pix_idx].B = 0;
    }
}


void overlap_removal_cuda(Context_gpu* ctx_gpu, framePacket** frameList) {
#if 1
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
#endif
    for(int i=0; i<numKinects; i++) {
        hipMemcpy(ctx_gpu->vertices[i], frameList[i]->vertices, sizeof(Point3fRGB) * Width_depth_HR * Height_depth_HR, hipMemcpyHostToDevice);
        hipMemcpy(ctx_gpu->depth[i], frameList[i]->data_d, sizeof(float) * Width_depth_HR * Height_depth_HR, hipMemcpyHostToDevice);
        hipMemcpy(ctx_gpu->vertices_shift[(i+1) % numKinects], ctx_gpu->vertices[i], sizeof(Point3fRGB) * Width_depth_HR * Height_depth_HR, hipMemcpyDeviceToDevice);
        hipMemcpy(ctx_gpu->depth_shift[(i+1) % numKinects], ctx_gpu->depth[i], sizeof(float) * Width_depth_HR * Height_depth_HR, hipMemcpyDeviceToDevice);
    }

    dim3 blocks(Width_depth_HR / 32, Height_depth_HR / 8);
    dim3 threads(32, 8);

    for(int i=0; i<numKinects; i++) {
        overlap_removal_kernel<<<blocks, threads>>>(ctx_gpu, ctx_gpu->vertices[i], ctx_gpu->vertices_shift[i], ctx_gpu->depth_shift[i], (i+1) % numKinects);
    }

    for(int i=0; i<numKinects; i++) {
        hipMemcpy(frameList[i]->vertices, ctx_gpu->vertices_shift[(i + numKinects - 1) % numKinects], sizeof(Point3fRGB) * Width_depth_HR * Height_depth_HR, hipMemcpyDeviceToHost);
        // hipMemcpy(frameList[i]->vertices, ctx_gpu->vertices[i], sizeof(Point3fRGB) * Width_depth_HR * Height_depth_HR, hipMemcpyDeviceToHost);
    }

#if 1
    hipEventRecord(end);
    hipEventSynchronize(end);
    float millisecond = 0;
    hipEventElapsedTime(&millisecond, start, end);
    printf("\t overlap_removal_cuda time = %f ms\n", millisecond);
#endif
}