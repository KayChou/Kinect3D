#include "hip/hip_runtime.h"
#include "cuda.cuh"

Context_gpu* create_context(Context* ctx_cpu) {
    hipSetDevice(0);

    Context_gpu *ctx_gpu;
    hipMallocManaged((void**)&ctx_gpu, sizeof(Context_gpu));
    hipMallocManaged((void**)&ctx_gpu->depth_out, 512 * 424 * sizeof(float));
    hipMalloc((void**)&ctx_gpu->mask, sizeof(float) * Width_depth_HR * Height_depth_HR);
    hipMallocManaged((void**)&ctx_gpu->matched_idx, sizeof(int) * Width_depth_HR * Height_depth_HR);
    hipMallocManaged((void**)&ctx_gpu->verts1_match, sizeof(Point3fRGB) * Width_depth_HR * Height_depth_HR);
    hipMallocManaged((void**)&ctx_gpu->verts2_match, sizeof(Point3fRGB) * Width_depth_HR * Height_depth_HR);
    hipMallocManaged((void**)&ctx_gpu->depth_match, sizeof(float) * Width_depth_HR * Height_depth_HR);

    for(int i=0; i<numKinects; i++) {
        hipMallocManaged((void**)&ctx_gpu->R[i], sizeof(float) * 9);
        hipMallocManaged((void**)&ctx_gpu->T[i], sizeof(float) * 3);
        hipMallocManaged((void**)&ctx_gpu->invR[i], sizeof(float) * 9);
        hipMallocManaged((void**)&ctx_gpu->vertices[i], sizeof(Point3fRGB) * Width_depth_HR * Height_depth_HR);
        hipMalloc((void**)&ctx_gpu->depth[i], sizeof(float) * Width_depth_HR * Height_depth_HR);

        ctx_gpu->color_params[i].Ar = 1;
        ctx_gpu->color_params[i].Br = 0;
        ctx_gpu->color_params[i].Ag = 1;
        ctx_gpu->color_params[i].Bg = 0;
        ctx_gpu->color_params[i].Ab = 1;
        ctx_gpu->color_params[i].Bb = 0;
    }
    ctx_gpu->width = Width_depth_HR;
    ctx_gpu->height = Height_depth_HR;
    ctx_gpu->x_ratio = Width_depth_HR / 512.f;
    ctx_gpu->y_ratio = Height_depth_HR / 424.f;
    
    return ctx_gpu;
}


void updata_context(Context_gpu *ctx_gpu, Context *ctx_cpu) {
    for(int i=0; i<numKinects; i++) {
        hipMemcpy(&ctx_gpu->R[i][0], &ctx_cpu->R[i][0][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][1], &ctx_cpu->R[i][0][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][2], &ctx_cpu->R[i][0][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][3], &ctx_cpu->R[i][1][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][4], &ctx_cpu->R[i][1][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][5], &ctx_cpu->R[i][1][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][6], &ctx_cpu->R[i][2][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][7], &ctx_cpu->R[i][2][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->R[i][8], &ctx_cpu->R[i][2][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->T[i][0], &ctx_cpu->T[i][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->T[i][1], &ctx_cpu->T[i][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->T[i][2], &ctx_cpu->T[i][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][0], &ctx_cpu->invR[i][0][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][1], &ctx_cpu->invR[i][0][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][2], &ctx_cpu->invR[i][0][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][3], &ctx_cpu->invR[i][1][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][4], &ctx_cpu->invR[i][1][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][5], &ctx_cpu->invR[i][1][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][6], &ctx_cpu->invR[i][2][0], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][7], &ctx_cpu->invR[i][2][1], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->invR[i][8], &ctx_cpu->invR[i][2][2], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->K[i].fx, &ctx_cpu->K[i].fx, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->K[i].fy, &ctx_cpu->K[i].fy, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->K[i].cx, &ctx_cpu->K[i].cx, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&ctx_gpu->K[i].cy, &ctx_cpu->K[i].cy, sizeof(float), hipMemcpyHostToDevice);
    }
}


__device__ void backforward_mapping(float *point, float *R, float *T)
{
    float res[3];
    res[0] = point[0] * R[0] + point[1] * R[1] + point[2] * R[2];
    res[1] = point[0] * R[3] + point[1] * R[4] + point[2] * R[5];
    res[2] = point[0] * R[6] + point[1] * R[7] + point[2] * R[8];

    point[0] = res[0] - T[0];
    point[1] = res[1] - T[1];
    point[2] = res[2] - T[2];
}


__global__ void overlap_removal_kernel(Context_gpu* ctx_gpu, Point3fRGB* left, float* depth_left, Point3fRGB* right, float* depth_right, int idx, float *depth_out) {
    int x_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int y_idx = threadIdx.y + blockIdx.y * blockDim.y;
    int pix_idx = x_idx + y_idx * blockDim.x * gridDim.x;
    depth_out[pix_idx] = 128;
    ctx_gpu->mask[pix_idx] = 0;

    float tempPoint[3];
    tempPoint[0] = left[pix_idx].X;
    tempPoint[1] = left[pix_idx].Y;
    tempPoint[2] = left[pix_idx].Z;

    backforward_mapping(tempPoint, ctx_gpu->invR[idx], ctx_gpu->T[idx]);

    if(tempPoint[2] <= 0) {
        return;
    }

    int x = (int)(tempPoint[0] * ctx_gpu->x_ratio / tempPoint[2] * ctx_gpu->K[idx].fx + ctx_gpu->K[idx].cx - 0.5);
    int y = (int)(tempPoint[1] * ctx_gpu->y_ratio / tempPoint[2] * ctx_gpu->K[idx].fy + ctx_gpu->K[idx].cy - 0.5);

    bool b_overlap = false;
    int index;

    if(x >= 0 && x < Width_depth_HR && y >= 0 && y < Height_depth_HR) {
        for(int i=0; i<patch_size; i++) {
            for(int j=0; j<patch_size; j++) {
                index = ((y - patch_size/2) + i) * Width_depth_HR + (x - patch_size/2) + j;
                if(index >= 0 && index < Width_depth_HR * Height_depth_HR && depth_right[index] > 0 && abs(tempPoint[2] * 1000 - depth_right[index]) < 20) {
                    b_overlap = true;
                }
            }
        }
    }

    if(b_overlap) {
        ctx_gpu->mask[pix_idx] = 1;
    }

    return;
}


__global__ void patch_based_removal_kernel(int *mask, Point3fRGB* verts, float *depth) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    int pix_idx = x + y * blockDim.x * gridDim.x;
    int index;
    int cnt = 0;

    for(int i=0; i<removal_det_region; i++) {
        for(int j=0; j<removal_det_region; j++) {
            index = ((y - patch_size/2) + i) * Width_depth_HR + (x - patch_size/2) + j;
            if(mask[index] == 1) {
                cnt++;
            }
        }
    }

    if(cnt > removal_det_K) {
        verts[pix_idx].X = 0;
        verts[pix_idx].Y = 0;
        verts[pix_idx].Z = 0;
        // verts[pix_idx].R = 255;
        // verts[pix_idx].G = 0;
        // verts[pix_idx].B = 0;
        depth[pix_idx] = 0;
    }
    return;
}


__global__ void SDC_filter(Context_gpu *ctx_gpu, Point3fRGB* verts, float *depth) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int pix_idx = x + y * blockDim.x * gridDim.x;

    if(verts[pix_idx].Z <= z_bbox_min || verts[pix_idx].Z >= z_bbox_max ||
       verts[pix_idx].X <= x_bbox_min || verts[pix_idx].X >= x_bbox_max ||
       verts[pix_idx].Y <= y_bbox_min || verts[pix_idx].Y >= y_bbox_max) {
        return;
    }

    if(x == 0 || y == 0 || x == ctx_gpu->width - 1 || y == ctx_gpu->height - 1) {
        return;
    }

    int c = pix_idx; // center
    int l = pix_idx - 1; // left
    int r = pix_idx + 1; // right
    int t = pix_idx - ctx_gpu->width; // top
    int d = pix_idx + ctx_gpu->width; // down

    if(abs(verts[c].X - verts[l].X) + abs(verts[c].Y - verts[l].Y) + abs(verts[c].Z - verts[l].Z) < SDC_filter_threashold && 
       abs(verts[c].X - verts[r].X) + abs(verts[c].Y - verts[r].Y) + abs(verts[c].Z - verts[r].Z) < SDC_filter_threashold && 
       abs(verts[c].X - verts[t].X) + abs(verts[c].Y - verts[t].Y) + abs(verts[c].Z - verts[t].Z) < SDC_filter_threashold && 
       abs(verts[c].X - verts[d].X) + abs(verts[c].Y - verts[d].Y) + abs(verts[c].Z - verts[d].Z) < SDC_filter_threashold ) {
    }
    else {
        verts[pix_idx].X = 0;
        verts[pix_idx].Y = 0;
        verts[pix_idx].Z = 0;
        verts[pix_idx].R = 0;
        verts[pix_idx].G = 0;
        verts[pix_idx].B = 0;
        depth[pix_idx] = 0;
    }
}


__global__ void isolate_points_filter(Context_gpu *ctx_gpu, Point3fRGB* verts, float *depth) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int pix_idx = x + y * blockDim.x * gridDim.x;

    if(verts[pix_idx].Z <= z_bbox_min || verts[pix_idx].Z >= z_bbox_max ||
       verts[pix_idx].X <= x_bbox_min || verts[pix_idx].X >= x_bbox_max ||
       verts[pix_idx].Y <= y_bbox_min || verts[pix_idx].Y >= y_bbox_max) {
        return;
    }

    int index;
    int cnt = 0;
    for(int i=0; i<3; i++) {
        for(int j=0; j<3; j++) {
            index = ((y - 1) + i) * Width_depth_HR + (x - 1) + j;
            if(index > 0 && index < Width_depth_HR * Height_depth_HR) {
                if(depth[index] == 0) {
                    cnt ++;
                }
            }
        }
    }

    if(cnt >= 3) {
        verts[pix_idx].X = 0;
        verts[pix_idx].Y = 0;
        verts[pix_idx].Z = 0;
        verts[pix_idx].R = 0;
        verts[pix_idx].G = 0;
        verts[pix_idx].B = 0;
        depth[pix_idx] = 0;
    }
}


__global__ void color_correction(Context_gpu *ctx_gpu, Point3fRGB* verts, int idx) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int pix_idx = x + y * blockDim.x * gridDim.x;

    verts[pix_idx].R = ctx_gpu->color_params[idx].Ar * verts[pix_idx].R + ctx_gpu->color_params[idx].Br;
    verts[pix_idx].G = ctx_gpu->color_params[idx].Ag * verts[pix_idx].G + ctx_gpu->color_params[idx].Bg;
    verts[pix_idx].B = ctx_gpu->color_params[idx].Ab * verts[pix_idx].B + ctx_gpu->color_params[idx].Bb;
} 


void overlap_removal_cuda(Context_gpu* ctx_gpu, framePacket** frameList, float* dpeth_out, Context *ctx_cpu) {
#if 0
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
#endif
    dim3 blocks(Width_depth_HR / 32, Height_depth_HR / 8);
    dim3 threads(32, 8);

    for(int i=0; i<numKinects; i++) {
        hipMemcpy(ctx_gpu->vertices[i], frameList[i]->vertices, sizeof(Point3fRGB) * Width_depth_HR * Height_depth_HR, hipMemcpyHostToDevice);
        hipMemcpy(ctx_gpu->depth[i], frameList[i]->data_d, sizeof(float) * Width_depth_HR * Height_depth_HR, hipMemcpyHostToDevice);
    }

    if(ctx_cpu->b_SDC_filter) {
        for(int i=0; i<numKinects; i++) {
            SDC_filter<<<blocks, threads>>>(ctx_gpu, ctx_gpu->vertices[i], ctx_gpu->depth[i]);
        }
    }

    if(ctx_cpu->b_isolate_filter) {
        for(int i=0; i<numKinects; i++) {
            isolate_points_filter<<<blocks, threads>>>(ctx_gpu, ctx_gpu->vertices[i], ctx_gpu->depth[i]);
        }
    }

    int idx;
    for(int i=0; i<numKinects; i++) {
        idx = (i + 1) % numKinects;
        overlap_removal_kernel<<<blocks, threads>>>(ctx_gpu, ctx_gpu->vertices[i], ctx_gpu->depth[i], ctx_gpu->vertices[idx], ctx_gpu->depth[idx], idx, ctx_gpu->depth_out);
        patch_based_removal_kernel<<<blocks, threads>>>(ctx_gpu->mask, ctx_gpu->vertices[i], ctx_gpu->depth[i]);
    }
    for(int i=0; i<numKinects; i++) {
        hipMemcpy(frameList[i]->vertices, ctx_gpu->vertices[i], sizeof(Point3fRGB) * Width_depth_HR * Height_depth_HR, hipMemcpyDeviceToHost);
    }
    hipMemcpy(dpeth_out, ctx_gpu->depth_out, sizeof(float) * 512 * 424, hipMemcpyDeviceToHost);

#if 0
    hipEventRecord(end);
    hipEventSynchronize(end);
    float millisecond = 0;
    hipEventElapsedTime(&millisecond, start, end);
    printf("\t overlap_removal_cuda time = %f ms\n", millisecond);
#endif
}